#include "hip/hip_runtime.h"
#include <CL/cl.h>
#include <hip/hip_runtime.h>

#define MAX_FILTER_SIZE 7
#define BLOCK_SIZE 16
#define LOCAL_OFFSET (MAX_FILTER_SIZE / 2)
#define PADDED_SIZE (BLOCK_SIZE + LOCAL_OFFSET * 2)

static inline int roundUpDiv(int number, int base) {
  return ((number + base - 1) / base);
}

__constant__ float filter[MAX_FILTER_SIZE * MAX_FILTER_SIZE];

__global__ void convolution(int imageWidth, int imageHeight, float *inputImage,
                            float *outputImage, int filterWidth) {
  __shared__ float sharedImage[PADDED_SIZE][PADDED_SIZE];

  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if (ix >= imageWidth || iy >= imageHeight) return;

  int lx = threadIdx.x;
  int ly = threadIdx.y;

  int halfFilterSize = filterWidth / 2;

  // copy image from global memory to local memory
  for (int i = -halfFilterSize; i <= halfFilterSize; i += halfFilterSize) {
    for (int j = -halfFilterSize; j <= halfFilterSize; j += halfFilterSize) {
      int x = ix + i, y = iy + j;
      if (x < 0 || x >= imageWidth || y < 0 || y >= imageHeight) continue;

      int localX = lx + i + LOCAL_OFFSET, localY = ly + j + LOCAL_OFFSET;
      sharedImage[localX][localY] = inputImage[y * imageWidth + x];
    }
  }

  __threadfence_block();

  int startX = -min(ix, halfFilterSize);
  int endX = min(imageWidth - ix - 1, halfFilterSize);
  int startY = -min(iy, halfFilterSize);
  int endY = min(imageHeight - iy - 1, halfFilterSize);

  float sum = 0.0f;
  for (int i = startX; i <= endX; ++i) {
    for (int j = startY; j <= endY; ++j) {
      int x = lx + i + LOCAL_OFFSET, y = ly + j + LOCAL_OFFSET;
      int fx = halfFilterSize + i, fy = halfFilterSize + j;
      sum += sharedImage[x][y] * filter[fy * filterWidth + fx];
    }
  }

  outputImage[iy * imageWidth + ix] = sum;
}

void hostFE(int filterWidth, float *hostFilter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage, cl_device_id *, cl_context *,
            cl_program *) {
  int imageSize = imageWidth * imageHeight * sizeof(float);
  int filterSize = filterWidth * filterWidth * sizeof(float);

  float *deviceInputImage;
  hipMalloc((void **)&deviceInputImage, imageSize);
  float *deviceOutputImage;
  hipMalloc((void **)&deviceOutputImage, imageSize);

  hipMemcpy(deviceInputImage, inputImage, imageSize, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(filter), hostFilter, filterSize);

  dim3 blockShape(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridShape(roundUpDiv(imageWidth, blockShape.x),
                 roundUpDiv(imageHeight, blockShape.y));

  convolution<<<gridShape, blockShape>>>(imageWidth, imageHeight,
                                         deviceInputImage, deviceOutputImage,
                                         filterWidth);

  hipDeviceSynchronize();

  hipMemcpy(outputImage, deviceOutputImage, imageSize, hipMemcpyDeviceToHost);

  // hipFree(deviceInputImage);
  // hipFree(deviceOutputImage);
}