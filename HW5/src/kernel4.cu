#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE_X 8
#define BLOCK_SIZE_Y 4

__device__ int mandel(float c_re, float c_im, int count) {
  if (c_re * c_re + c_im * c_im <= .25f * .25f) {
    return count;
  }

  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX,
                             float stepY, int maxIteration, int resX,
                             int* dest) {
  // To avoid error caused by the floating number, use the following pseudo code
  //
  // float x = lowerX + thisX * stepX;
  // float y = lowerY + thisY * stepY;

  int thisX = blockIdx.x * blockDim.x + threadIdx.x;
  int thisY = blockIdx.y * blockDim.y + threadIdx.y;

  float x = lowerX + thisX * stepX;
  float y = lowerY + thisY * stepY;

  int index = thisY * resX + thisX;
  dest[index] = mandel(x, y, maxIteration);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int sizeInBytes = sizeof(int) * resX * resY;

  int* deviceImg;
  hipHostRegister(img, sizeInBytes, hipHostRegisterMapped);
  hipHostGetDevicePointer((void**)&deviceImg, img, 0);

  dim3 blockShape(BLOCK_SIZE_X, BLOCK_SIZE_Y);
  dim3 gridShape(qresX / blockShape.x, resY / blockShape.y);
  mandelKernel<<<gridShape, blockShape>>>(lowerX, lowerY, stepX, stepY,
                                          maxIterations, resX, deviceImg);

  hipDeviceSynchronize();

  hipHostUnregister(img);

  hipMemcpy(img, deviceImg, sizeInBytes, hipMemcpyDeviceToHost);

  hipFree(deviceImg);
}
