#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GROUP_SIZE 4

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX,
                             float stepY, int maxIteration, size_t pitch,
                             int resX, int resY, int* dest) {
  // To avoid error caused by the floating number, use the following pseudo code
  //
  // float x = lowerX + thisX * stepX;
  // float y = lowerY + thisY * stepY;

  int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
  int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * GROUP_SIZE;

  for (int currentY = thisY; currentY < thisY + GROUP_SIZE; ++currentY) {
    if (currentY >= resY) break;
    float y = lowerY + currentY * stepY;
    int* row = (int*)((char*)dest + currentY * pitch);

    for (int currentX = thisX; currentX < thisX + GROUP_SIZE; ++currentX) {
      if (currentX >= resX) continue;
      float x = lowerX + currentX * stepX;
      row[currentX] = mandel(x, y, maxIteration);
    }
  }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int rowSize = sizeof(int) * resX;
  int sizeInBytes = sizeof(int) * resX * resY;

  int* deviceImg;
  size_t pitch;
  hipMallocPitch((void**)&deviceImg, &pitch, rowSize, resY);

  dim3 blockShape(BLOCK_SIZE, BLOCK_SIZE);

  int numBlocksX = ceil(resX / float(blockShape.x * GROUP_SIZE));
  int numBlocksY = ceil(resX / float(blockShape.x * GROUP_SIZE));

  dim3 gridShape(numBlocksX, numBlocksY);
  mandelKernel<<<gridShape, blockShape>>>(lowerX, lowerY, stepX, stepY,
                                          maxIterations, pitch, resX, resY,
                                          deviceImg);

  hipDeviceSynchronize();

  int* hostImg;
  hipHostAlloc((void**)&hostImg, sizeInBytes, hipHostMallocDefault);

  hipMemcpy2D(hostImg, rowSize, deviceImg, pitch, rowSize, resY,
               hipMemcpyDeviceToHost);
  memcpy(img, hostImg, sizeInBytes);

  hipHostFree(hostImg);
  hipFree(deviceImg);
}
