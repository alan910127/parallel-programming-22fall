#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GROUP_SIZE 4

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX,
                             float stepY, int maxIteration, int resX,
                             int* dest) {
  // To avoid error caused by the floating number, use the following pseudo code
  //
  // float x = lowerX + thisX * stepX;
  // float y = lowerY + thisY * stepY;

  int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
  int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * GROUP_SIZE;

  for (int currentX = thisX; currentX < thisX + GROUP_SIZE; ++currentX) {
    for (int currentY = thisY; currentY < thisY + GROUP_SIZE; ++currentY) {
      float x = lowerX + currentX * stepX;
      float y = lowerY + currentY * stepY;

      float* row = (float*)((char*)dest + currentY * pitch);
      row[currentX] = mandel(x, y, maxIteration);
    }
  }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int rowSize = sizeof(int) * resX;
  int sizeInBytes = rowSize * resY;

  int* deviceImg;
  size_t pitch;
  hipMallocPitch((void**)&deviceImg, &pitch, rowSize, resY);

  dim3 blockShape(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridShape(resX / (blockShape.x * GROUP_SIZE),
                 resY / (blockShape.y * GROUP_SIZE));
  mandelKernel<<<gridShape, blockShape>>>(
      lowerX, lowerY, stepX, stepY, maxIterations, resX, pitch, deviceImg);

  int* hostImg;
  hipHostAlloc((void**)&hostImg, sizeInBytes, hipHostMallocDefault);

  hipMemcpy2D(hostImg, rowSize, deviceImg, pitch, rowSize, resY,
               hipMemcpyDeviceToHost);
  memcpy(img, hostImg, sizeInBytes);

  hipHostFree(hostImg);
  hipFree(deviceImg);
}
