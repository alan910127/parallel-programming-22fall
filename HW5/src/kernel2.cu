#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX,
                             float stepY, int maxIteration, size_t pitch,
                             int* dest) {
  // To avoid error caused by the floating number, use the following pseudo code
  //
  // float x = lowerX + thisX * stepX;
  // float y = lowerY + thisY * stepY;

  int thisX = blockIdx.x * blockDim.x + threadIdx.x;
  int thisY = blockIdx.y * blockDim.y + threadIdx.y;

  float x = lowerX + thisX * stepX;
  float y = lowerY + thisY * stepY;

  int* row = (int*)((char*)dest + thisY * pitch);
  row[thisX] = mandel(x, y, maxIteration);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int rowSize = sizeof(int) * resX;
  int sizeInBytes = rowSize * resY;

  int* deviceImg;
  size_t pitch;
  hipMallocPitch((void**)&deviceImg, &pitch, rowSize, resY);

  dim3 blockShape(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridShape(resX / blockShape.x, resY / blockShape.y);
  mandelKernel<<<gridShape, blockShape>>>(lowerX, lowerY, stepX, stepY,
                                          maxIterations, pitch, deviceImg);

  hipDeviceSynchronize();

  int* hostImg;
  hipHostAlloc((void**)&hostImg, sizeInBytes, hipHostMallocDefault);

  hipMemcpy2D(hostImg, rowSize, deviceImg, pitch, rowSize, resY,
               hipMemcpyDeviceToHost);
  memcpy(img, hostImg, sizeInBytes);

  hipHostFree(hostImg);
  hipFree(deviceImg);
}
