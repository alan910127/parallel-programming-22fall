#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX,
                             float stepY, int maxIteration, int resX,
                             int* dest) {
  // To avoid error caused by the floating number, use the following pseudo code
  //
  // float x = lowerX + thisX * stepX;
  // float y = lowerY + thisY * stepY;

  int thisX = blockIdx.x * blockDim.x + threadIdx.x;
  int thisY = blockIdx.y * blockDim.y + threadIdx.y;

  float x = lowerX + thisX * stepX;
  float y = lowerY + thisY * stepY;

  int index = thisY * resX + thisX;
  dest[index] = mandel(x, y, maxIteration);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int sizeInBytes = sizeof(int) * resX * resY;

  int* deviceImg;
  hipMalloc((void**)&deviceImg, sizeInBytes);

  dim3 blockShape(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridShape(resX / blockShape.x, resY / blockShape.y);
  mandelKernel<<<gridShape, blockShape>>>(lowerX, lowerY, stepX, stepY,
                                          maxIterations, resX, deviceImg);

  hipDeviceSynchronize();

  int* hostImg = (int*)malloc(sizeInBytes);
  hipMemcpy(hostImg, deviceImg, sizeInBytes, hipMemcpyDeviceToHost);
  memcpy(img, hostImg, sizeInBytes);

  free(hostImg);
  hipFree(deviceImg);
}
